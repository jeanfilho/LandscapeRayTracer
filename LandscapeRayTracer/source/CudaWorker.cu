#include "hip/hip_runtime.h"
#pragma once

#include "CudaWorker.h"

__device__ glm::fvec3* d_pixel_array;

__global__
void castRay()
{

}

void CudaWorker::par_castRay(glm::fvec3* pixel_array, const Camera* cam, const Grid<Grid<PointData*>*>* grid, int window_height, int window_width)
{
	if (hipMalloc(&d_pixel_array, window_height*window_width * 3 * sizeof(float)) != hipSuccess)
	{
		cout << "Error allocating memory in device" << endl;
	}
	hipMemcpy(d_pixel_array, pixel_array, window_height*window_width * 3 * sizeof(float), hipMemcpyHostToDevice);

	castRay << <4, 4 >> >();

	hipFree(d_pixel_array);
}